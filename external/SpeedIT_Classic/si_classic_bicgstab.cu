#include "hip/hip_runtime.h"
/*
 * si_classic.cu - part of the SpeedIT Classic toolkit
 * Copyright (C) 2009 - 2010 Vratis
 * email: support@vratis.com
 * 
 * SpeedIT Classic toolkit is a free software: you can redistribute it and/or
 * modify it under the terms of the GNU General Public License as published 
 * by the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * SpeedIT Classic library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 * See the GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License along
 * with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "hip/hip_runtime.h"
#include "hipblas.h"
#include <iostream>
#include <limits>

#include "si_classic.h"

using namespace std ;


const int BLOCK_SIZE = 256;

////////////////////////////////////////////////////////////////////////////////
//
//                          Internal functions
//
////////////////////////////////////////////////////////////////////////////////

//------------------------------------------------------------------------------
//
//                              Utilities
//
//------------------------------------------------------------------------------

//
//  Function translates cublas error codes to simple string message.
//
static const char* 
cublasErrStr(int err_code)
{
  switch(err_code)
  {
    case HIPBLAS_STATUS_SUCCESS         :  
      return "HIPBLAS_STATUS_SUCCESS"          ;
    case HIPBLAS_STATUS_NOT_INITIALIZED :  
      return "HIPBLAS_STATUS_NOT_INITIALIZED"  ;
    case HIPBLAS_STATUS_ALLOC_FAILED    :  
      return "HIPBLAS_STATUS_ALLOC_FAILED"     ;
    case HIPBLAS_STATUS_INVALID_VALUE   :  
      return "HIPBLAS_STATUS_INVALID_VALUE"    ;
    case HIPBLAS_STATUS_ARCH_MISMATCH   :  
      return "HIPBLAS_STATUS_ARCH_MISMATCH"    ;
    case HIPBLAS_STATUS_MAPPING_ERROR   :  
      return "HIPBLAS_STATUS_MAPPING_ERROR"    ;
    case HIPBLAS_STATUS_EXECUTION_FAILED:  
      return "HIPBLAS_STATUS_EXECUTION_FAILED" ;
    case HIPBLAS_STATUS_INTERNAL_ERROR  :  
      return "HIPBLAS_STATUS_INTERNAL_ERROR"   ;
    default                            :
      return "Unknown CUBLAS ERROR"           ;
  } ;
} ;

//
//  Function mainly used to call CUDA routines and check, if everything
//  gone well. 
//
static inline void 
cudaCall( 
                hipError_t   err, 
          const char      * msg 
        )
{
  if( hipSuccess != err) 
  { 
    cerr << msg << " : "              ;
    cerr << hipGetErrorString( err ) ;
    cerr << "\n"                      ;
    exit(-1)                          ;
  } ;
} ;

//
//  Function mainly used to call cublas routines and check, if everything
//  gone well. 
//
static inline void 
cublasCall( 
                  hipblasStatus_t   err, 
            const char         * msg 
          )
{
  if (HIPBLAS_STATUS_SUCCESS != err)
  {
    cerr << msg << " : "        ;
    cerr << cublasErrStr( err ) ;
    cerr << "\n"                ;
    exit(-1)                    ;
  } ;
} ;

//------------------------------------------------------------------------------
//
//                            CUDA kernels
//
//------------------------------------------------------------------------------

//
//  GPU kernel to multiply sparse matrix in CSR format by dense vector
//
static __global__ void
KERNEL_crs_multiply(        int     n_rows     ,  // numer of matrix rows
                      const float * vals       ,  // matrix nonzeros
                      const int   * col_idx    ,  // column indices fo nonzeros
                      const int   * row_offset ,  // positions of rows begin, start from 0
                      const float * X          ,  // vector, by which matrix is multiplied
                            float * R             // result 
                    )
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= n_rows) 
    return;

  float sum = 0.0 ;

  for (int k = row_offset[ idx ]; k < row_offset[idx + 1]; k++)
  {
    sum += vals[ k ] * X[ col_idx[ k ] ];
  }

  R[ idx ] = sum ;
} ;

//
//  GPU kernel for simple vector operation
//
//      R = B - R
//
//  where R and B are dense vectors. Kernel is used in CG solver.
//
static __global__ void 
KERNEL_replace_with_residual (        int    n_rows , // vector sizes
                               const float * B      , // pointers to vectors
                                     float * R        //
                              )
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x ;

  if (idx >= n_rows)  
    return;

  R[idx] = B[idx] - R[idx] ;
} ;

//
//  GPU kernel for simple vector operation
//
//      S = R + alpha * V
//
//  where R, S and V are dense vectors. Kernel is used in CG solver.
//
static __global__ void
KERNEL_smul_vadd(
                        int     n_rows ,
                  const float * R      , 
                  const float * V      , 
                        float   alpha  , 
                        float * S       
                )
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x ;

  if (idx >= n_rows) return ;

  S[ idx ] = R[ idx ] + alpha * V[ idx ] ;
} ;

//
//  GPU kernel for simple vector operation
//
//      X = X + alpha * P
//
//  where X and P are dense vectors. Kernel is used in CG solver.
//
static __global__ void
KERNEL_add_to_x (
                        int     n_rows ,
                        float   alpha  , 
                  const float * P      ,
                        float * X      
                )
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= n_rows)  // TODO: tu byl error, poprawic w trunku
    return;

  X[ idx ] += alpha * P[ idx ];
} ;

//------------------------------------------------------------------------------
//
//                          Computational routines
//
//------------------------------------------------------------------------------

//
//  Wrapper for hipblasSnrm2() function
//
static inline float 
norm(
      const float * v, 
      const int     size
    )
{
  float result = hipblasSnrm2(size, v, 1)      ;
  cublasCall(cublasGetError(), "norm FAILED") ;

  return result;
} ;

//
//  Wrapper for hipblasSdot() function
//
static inline float
dot_product (
                    int n_rows ,
              const float* v   ,
              const float* w
            )
{
  float result =  hipblasSdot(n_rows, v, 1, w, 1)     ;
  cublasCall(cublasGetError(), "dot_product FAILED") ;

  return result ;
} ;

//
//  Function used in CG solver
//
static void
calc_residual(      int     n_rows ,
              const float * vals   ,
              const int   * c_idx  ,
              const int   * r_idx  ,
              const float * B      ,
              const float * X      ,
                    float * R
              )
{
  const int size = n_rows ;
  int numBlocks = size / BLOCK_SIZE + (size % BLOCK_SIZE == 0 ? 0 : 1) ;
  dim3 dimGrid(numBlocks) ;
  dim3 dimBlock(BLOCK_SIZE) ;


  sicl_gscsrmv(n_rows, vals, c_idx, r_idx, X, R);  // R = A*X;

  // R = B - R
  KERNEL_replace_with_residual <<<dimGrid, dimBlock>>> (n_rows, B, R) ;  

  hipDeviceSynchronize() ;
  cudaCall (hipGetLastError(), "KERNEL_replace_with_residual FAILED") ;
} ;

//
//  S = R + alpha * V
//
static void
smul_vadd(  
                  int     n_rows ,
            const float * R      ,
            const float * V      ,
                  float   alpha  ,
                  float * S    
         )
{
  const int size = n_rows;
  int numBlocks = size / BLOCK_SIZE + (size % BLOCK_SIZE == 0 ? 0 : 1) ;
  dim3 dimGrid(numBlocks) ;
  dim3 dimBlock(BLOCK_SIZE) ;

  KERNEL_smul_vadd<<<dimGrid,dimBlock>>>( n_rows, R, V, alpha, S) ;

  hipDeviceSynchronize() ;
  cudaCall (hipGetLastError(), "KERNEL_smul_vadd FAILED") ;
} ;

//
//  X = X + alpha * P
//
static void
add_to_x(
                int     n_rows ,
                float   alpha  ,
          const float * P      ,
                float * X
        )
{
  const int size = n_rows ;
  int numBlocks = size / BLOCK_SIZE + (size % BLOCK_SIZE == 0 ? 0 : 1);
  dim3 dimGrid(numBlocks);
  dim3 dimBlock(BLOCK_SIZE);

  KERNEL_add_to_x <<<dimGrid,dimBlock>>> (n_rows, alpha, P, X);

  hipDeviceSynchronize();
  cudaCall (hipGetLastError(), "KERNEL_add_to_x FAILED") ;
} ;


////////////////////////////////////////////////////////////////////////////////
//
//                            Exported functions
//
////////////////////////////////////////////////////////////////////////////////


//
//  Function computes sparse matrix by dense vector multiplication
//
//            A * X = R
//
//  where A - matrix, X and R - vectors. Computation is done with
//  GPU.
//
int 
sicl_gscsrmv(       
                int     n_rows ,  // numer of matrix rows
          const float * vals   ,  // matrix nonzeros
          const int   * c_idx  ,  // column indices fo nonzeros            
          const int   * r_idx  ,  // positions of rows begin, start from 0 
          const float * X      ,  // vector, by which matrix is multiplied
                float * R         // result 
            )
{
  static const int size = n_rows;
  static int numBlocks = size / BLOCK_SIZE + (size % BLOCK_SIZE == 0 ? 0 : 1);
  static dim3 dimGrid(numBlocks); 
  static dim3 dimBlock(BLOCK_SIZE);
  
  KERNEL_crs_multiply<<<dimGrid,dimBlock>>> (
                                              n_rows ,
                                              vals   ,
                                              c_idx  ,
                                              r_idx  ,
                                              X      ,
                                              R
                                            ) ;
  hipDeviceSynchronize();
  cudaCall (hipGetLastError(), "KERNEL_crs_multiply FAILED") ;

  return 0 ;
} ;

//
//  Function solves with preconditioned Conjugate Gradient (CG) method system of
//  linear algebraic equations
//
//      A * X = B
//
//  where A is a sparse matrix and X and B are dense vectors.
//
int 
sicl_gscsrcg(     int            n_rows  , 
            const float        * vals    , 
            const int          * c_idx   , 
            const int          * r_idx   , 
                  float        * X       , 
            const float        * B       , 
                  PRECOND_TYPE   precond , // Only for compatibility 
                  int          * n_iter  , 
                  float        * epsilon
             ) 
{
  int result = -1 ;
  const float almost_zero = numeric_limits<float>::min(); 

  // Notation, algorithm: see Barlett et al, p. 13
  float rho_1;      // \rho_{i-1}
  float rho_2 = 0;  // \rho_{i-2}   
  float alpha = 0;  // \alpha{i}    
  float beta;       // \beta_{i-1}

  float norm_b = norm(B, n_rows) ;

  if (norm_b < almost_zero) // if B == 0
  {
    norm_b = 1.0 ;
  }

  float *R, *P, *Z, *Q ;
  
  // residuals
  cudaCall(hipMalloc ((void**)(&R), n_rows*sizeof(float)), "hipMalloc failed for R") ;
  // search directions
  cudaCall(hipMalloc ((void**)(&P), n_rows*sizeof(float)), "hipMalloc failed for P") ;
  // solution of A * Z = R
  cudaCall(hipMalloc ((void**)(&Z), n_rows*sizeof(float)), "hipMalloc failed for Z") ;
  cudaCall(hipMalloc ((void**)(&Q), n_rows*sizeof(float)), "hipMalloc failed for Q") ;

  float* pZ = R ; 

  // R = B - AX;  (Line 1 at Barlett's algorithm)
  calc_residual(n_rows, vals, c_idx, r_idx, B, X, R) ;

  float residuum = norm(R, n_rows) / norm_b ;

  if (residuum < *epsilon)  // if the trial solution satisfies the equation...
  {
    *n_iter = 0 ;
    result  = 0 ;
  } 
  else
  {
    for (int niter = 1 ; niter <= *n_iter ; niter++)
    {
      rho_1 = dot_product(n_rows, R, pZ) ;

      if (1 == niter) {
        // p^1 = z^0;   Barlett: line 6
        cudaCall(hipMemcpy(P, pZ, n_rows*sizeof(float), hipMemcpyDeviceToDevice), 
            "hipMemcpy P = pz FAILED" ) ;
      } else {
        beta = (rho_1/rho_2) ;
        smul_vadd(n_rows, pZ, P, beta, P) ;             // P = Z + beta * P
      } ;

      sicl_gscsrmv(n_rows, vals, c_idx, r_idx, P, Q) ; // Q = A*P
      alpha = rho_1 / dot_product(n_rows, P, Q)  ;

      add_to_x(n_rows,  alpha, P, X) ;         // X^i = X^{i-1} + alpha_i * p^i
      add_to_x(n_rows, -alpha, Q, R) ;
      rho_2 = rho_1 ;

      residuum = norm(R, n_rows) / norm_b ;

      if (residuum < *epsilon) // iteration succeeded
      {
        *n_iter = niter ;
        result  = 0 ;
        break ;
      } ;
    } ;
  } ;

  *epsilon = residuum ;

  cudaCall(hipFree (Q), "hipFree failed for Q") ;
  cudaCall(hipFree (R), "hipFree failed for R") ;
  cudaCall(hipFree (P), "hipFree failed for P") ;
  cudaCall(hipFree (Z), "hipFree failed for Z") ;

  return result ;
} ;



static __global__ void
KERNEL_smul_vadd3(  
                  int     n_rows ,
            const float * R      ,
            const float * V      ,
            const float * W	 ,
                  float   alpha  ,
                  float	  beta, 
                  float * S    
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x ;

  if (idx >= n_rows) return ;

  S[ idx ] = R[ idx ] + alpha * V[ idx ] + beta * W[ idx ] ;
} ;



static void
smul_vadd3(  
                  int     n_rows ,
            const float * R      ,
            const float * V      ,
            const float * W	 ,
                  float   alpha  ,
                  float	  beta, 
                  float * S    
         )
{
  const int size = n_rows;
  int numBlocks = size / BLOCK_SIZE + (size % BLOCK_SIZE == 0 ? 0 : 1) ;
  dim3 dimGrid(numBlocks) ;
  dim3 dimBlock(BLOCK_SIZE) ;

  KERNEL_smul_vadd3<<<dimGrid,dimBlock>>>( n_rows, R, V,W, alpha,beta, S) ;

  hipDeviceSynchronize() ;
  cudaCall (hipGetLastError(), "KERNEL_smul_vadd FAILED") ;
} ;


/* BICGSTAB - stabilized BiConjugate Gradients algorithm
 */
int bicgstab_kernel(     int            n_rows  , 
				    const float        * vals    , 
				    const int          * c_idx   , 
				    const int          * r_idx   , 
					  float        * X       , 
				    const float        * B       , 
					  PRECOND_TYPE   precond , // Only for compatibility 
					  int          * n_iter  , 
					  float        * epsilon
             			) 
{

  float norm_b; /* 2 nrm of residual */ 
  const float almost_zero = numeric_limits<float>::min(); 

  /* Check for all zero right hand side vector => all zero solution */
  
  norm_b = norm(B, n_rows) ;	/* Norm of rhs vector, b */
  if (norm_b < almost_zero) {	/* if rhs vector is all zeros */
    norm_b = 1.0;
  }
  
  float *R, *Rhat, *V, *P, *T ;
  
  // residuals
  cudaCall(hipMalloc ((void**)(&R), n_rows*sizeof(float)), "hipMalloc failed for R") ;
  cudaCall(hipMalloc ((void**)(&Rhat), n_rows*sizeof(float)), "hipMalloc failed for Rhat") ;
  // search directions
  cudaCall(hipMalloc ((void**)(&V), n_rows*sizeof(float)), "hipMalloc failed for V") ;
  // solution of A * Z = R
  cudaCall(hipMalloc ((void**)(&P), n_rows*sizeof(float)), "hipMalloc failed for P") ;
  cudaCall(hipMalloc ((void**)(&T), n_rows*sizeof(float)), "hipMalloc failed for T") ;
  
  float omega_1=1.0, alpha=1.0, rho_1=1.0, omega, rho, beta;
  
   // R = B - AX;  (Line 1 at Barlett's algorithm)
  calc_residual(n_rows, vals, c_idx, r_idx, B, X, R) ;

  float residuum = norm(R, n_rows) / norm_b ;

  if (residuum < *epsilon)  // if the trial solution satisfies the equation...
  {
    *n_iter = 0 ;
    result  = 0 ;
  }
  else
  {
    cudaCall(hipMemcpy(Rhat, R, n_rows*sizeof(float), hipMemcpyDeviceToDevice); //r=rhat
    
    for (int niter = 1 ; niter <= *n_iter ; niter++)
    {
      rho = dot_product(n_rows, Rhat, R) ;
      beta = (rho/rho_1)*(alpha/omega_1);
      
      /*if (1 == niter) {
        // p^1 = z^0;   Barlett: line 6
        cudaCall(hipMemcpy(P, Rhat, n_rows*sizeof(float), hipMemcpyDeviceToDevice), 
            "hipMemcpy P = pz FAILED" ) ;
      } else {
        beta = (rho_1/rho_2) ;
        smul_vadd(n_rows, pZ, P, beta, P) ;             // P = Z + beta * P
      } ;*/
      
      if ( 1== niter){
      		cudaCall(hipMemcpy(P, R, n_rows*sizeof(float), hipMemcpyDeviceToDevice), 
            "hipMemcpy P = R FAILED" ) ;
      }
      else{
      		
      		smul_vadd3(n_rows, R, P,V,beta,-beta*omega_1, P) ;  // P = R+ beta*P - (beta*omega_1)*V 
      }
      
      sicl_gscsrmv(n_rows, vals, c_idx, r_idx, P, V) ; // V = A*P
      alpha = rho / dot_product(n_rows, Rhat, V)  ;
      smul_vadd(n_rows, R, V,-alpha, S) ;             // S = R - alpha * V
      sicl_gscsrmv(n_rows, vals, c_idx, r_idx, S, T) ; // T = A*S	
      
      omega = dot_product(n_rows, T, S)/dot_product(n_rows, T,T);
      smul_vadd3(n_rows, X, P, S,alpha,omega, S) ;
      smul_vadd(n_rows, S, T,-omega, R) ;             // S = R - alpha * V
      
      
      //add_to_x(n_rows,  alpha, P, X) ;         // X^i = X^{i-1} + alpha_i * p^i
      //add_to_x(n_rows, -alpha, Q, R) ;
      
      omega_1 = omega;
      rho_1 = rho;

      residuum = norm(R, n_rows) / norm_b ;

      if (residuum < *epsilon) // iteration succeeded
      {
        *n_iter = niter ;
        result  = 0 ;
        break ;
      } ;
    } ;
  } ;

  *epsilon = residuum ;


  return(0);

}


